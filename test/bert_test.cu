#include "hip/hip_runtime.h"
#include "bert/bert.cuh"
#include "nn/nexus_utility.cuh"

#include <precompiled/catch2_includes.h>

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;
using namespace nexus;

constexpr double MAX_RTOL=1e-3;
constexpr double MAX_ATOL=1e-2;

torch::Tensor random_tensor(torch::IntArrayRef size, double min, double max) {
    return torch::rand(size, torch::kDouble) * (max - min) + min;   
}

TEST_CASE("BERT Components") {
    auto poly_modulus_degree = 1ULL << 16;
    double scale = pow(2.0, 40);
    EncryptionParameters parms(scheme_type::ckks);
    
    vector<int> coeff_modulus{60};
    for (int i=0; i<22; i++)
        coeff_modulus.push_back(40);
    coeff_modulus.push_back(60);

    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, coeff_modulus));

    auto context = std::make_shared<PhantomContext>(parms);
    auto secret_key = std::make_shared<PhantomSecretKey>(*context);
    auto public_key = std::make_shared<PhantomPublicKey>(secret_key->gen_publickey(*context));
    auto relin_keys = std::make_shared<PhantomRelinKey>(secret_key->gen_relinkey(*context));
    auto galois_keys = std::make_shared<PhantomGaloisKey>(secret_key->create_galois_keys(*context));

    auto encoder = std::make_shared<PhantomCKKSEncoder>(*context);

    auto ckks_evaluator = std::make_shared<CKKSEvaluator>(context, public_key, secret_key, encoder, relin_keys, galois_keys, scale);

    SECTION("Attention") {
        BertAttention attention(ckks_evaluator);

        torch::Tensor input = random_tensor({128, 768}, -0.5, 0.5);
        auto gt_output = attention.forward(input.to(torch::kFloat));

        auto packed_input = row_pack_128x768(input);
        std::vector<PhantomCiphertext> input_ct;
        for (auto &inp : packed_input) {
            input_ct.push_back(CKKSEncrypt(inp, ckks_evaluator));
        }

        attention.pack_weights();

        auto out = attention.forward(input_ct);

        std::vector<torch::Tensor> decrypted_out;
        for (auto &o : out) {
            auto tensor_out = tensor_from_vector(CKKSDecrypt(o, ckks_evaluator), {2, 128, 128});
            decrypted_out.push_back(tensor_out.index({0}));
            decrypted_out.push_back(tensor_out.index({1}));
        }
        torch::Tensor attn_output = torch::concat(decrypted_out, -1);

        CHECK(torch::allclose(attn_output.to(torch::kFloat), gt_output, MAX_RTOL, MAX_ATOL));
    }
    
    SECTION("MLP") {
        BertMLP mlp(ckks_evaluator);

        torch::Tensor input = random_tensor({128, 768}, -0.5, 0.5);
        torch::Tensor gt_output = mlp.forward(input.to(torch::kFloat));

        auto packed_input = row_pack_128x768(input);
        std::vector<PhantomCiphertext> input_ct;
        for (auto &inp : packed_input) {
            input_ct.push_back(CKKSEncrypt(inp, ckks_evaluator));
        }

        mlp.pack_weights();

        auto out = mlp.forward(input_ct);

        std::vector<torch::Tensor> decrypted_out;
        for (auto &o : out) {
            auto tensor_out = tensor_from_vector(CKKSDecrypt(o, ckks_evaluator), {2, 128, 128});
            decrypted_out.push_back(tensor_out.index({0}));
            decrypted_out.push_back(tensor_out.index({1}));
        }
        torch::Tensor output = torch::concat(decrypted_out, -1);

        CHECK(torch::allclose(output.to(torch::kFloat), gt_output, MAX_RTOL, MAX_ATOL));
    }
}